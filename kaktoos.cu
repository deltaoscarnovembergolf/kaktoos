#include "hip/hip_runtime.h"
#define GRID_SIZE (1LL << 24)
#define BLOCK_SIZE 512
#define CHUNK_SIZE (GRID_SIZE / BLOCK_SIZE)
#define RNG_MUL 25214903917ULL
#define RNG_ADD 11ULL
#define RNG_MASK ((1ULL << 48) - 1)

#ifndef CACTUS_HEIGHT
#define CACTUS_HEIGHT 7
#endif


#include <chrono>
#include <cstdint>
#include <mutex>
#include <thread>
#include <hip/hip_runtime.h>

#ifdef BOINC
  #include "boinc_api.h"
#if defined _WIN32 || defined _WIN64
  #include "boinc_win.h"
#endif
#endif

__device__ unsigned long long block_add_gpu[BLOCK_SIZE + 1];
__device__ unsigned long long block_mul_gpu[BLOCK_SIZE + 1];
__device__ unsigned long long chunk_add_gpu[CHUNK_SIZE + 1];
__device__ unsigned long long chunk_mul_gpu[CHUNK_SIZE + 1];
__device__ int32_t FLOOR_LEVEL;

__device__ inline int32_t next(uint32_t *random, uint32_t *index, int bits)
{
	return (random[(*index)++] >> (32 - bits));
}

__device__ inline int32_t next_int(uint32_t *random, uint32_t *index, int32_t bound)
{
	int32_t bits, value;
	do {
		bits = next(random, index, 31);
		value = bits % bound;
	} while (bits - value + (bound - 1) < 0);
	return value;
}

__device__ inline int32_t next_int_unknown(uint32_t *random, uint32_t *index, int32_t bound)
{
	if ((bound & -bound) == bound) {
		return (int32_t) ((bound * (unsigned long long) next(random, index, 31)) >> 31);
	} else {
		return next_int(random, index, bound);
	}
}

__device__ inline uint8_t extract(const uint32_t *heightmap, uint16_t pos)
{
	return ((heightmap[pos >> 3] >> ((pos & 7) << 2)) & 15) + FLOOR_LEVEL;
}

__device__ inline void increase(uint32_t *heightmap, uint16_t pos, uint8_t addend)
{
	heightmap[pos >> 3] += addend << ((pos & 7) << 2);
}

__global__ void crack(unsigned long long seed, unsigned long long *out, unsigned long long *out_n)
{
	__shared__ uint32_t random[BLOCK_SIZE + 1024];
	__shared__ uint32_t skip_index[BLOCK_SIZE + 1024 - 100];
	__shared__ uint32_t skip_first[BLOCK_SIZE + 1024 - 102];
	__shared__ uint32_t skip_always[BLOCK_SIZE + 1024 - 102];
	__shared__ uint32_t floor_skip[BLOCK_SIZE + 1024 - 102];
	__shared__ uint8_t floor_terrain[BLOCK_SIZE + 1024 - 102];
	__shared__ uint32_t offset_skip[BLOCK_SIZE + 1024 - 4];
	__shared__ uint8_t offset_height[BLOCK_SIZE + 1024 - 4];
	uint32_t heightmap[128];
	uint32_t random_index;

	seed = (seed * chunk_mul_gpu[blockIdx.x] + chunk_add_gpu[blockIdx.x]) & RNG_MASK;
	seed = (seed * block_mul_gpu[threadIdx.x] + block_add_gpu[threadIdx.x]) & RNG_MASK;
	unsigned long long seed2 = seed;
	seed = ((seed - 11ULL) * 246154705703781ULL) & RNG_MASK;
	random[threadIdx.x + BLOCK_SIZE * 0] = (uint32_t) (seed2 >> 16);
	for (int i = threadIdx.x + BLOCK_SIZE; i < BLOCK_SIZE + 1024; i += BLOCK_SIZE) {
		seed2 = (seed2 * block_mul_gpu[BLOCK_SIZE] + block_add_gpu[BLOCK_SIZE]) & RNG_MASK;
		random[i] = (uint32_t) (seed2 >> 16);
	}
	for (int i = 0; i < 128; i++) {
		heightmap[i] = 0;
	}
	__syncthreads();

	for (int i = threadIdx.x; i < BLOCK_SIZE + 1024 - 4; i += BLOCK_SIZE) {
		random_index = i;
		uint8_t offset = next_int_unknown(random, &random_index, next_int(random, &random_index, 3) + 1) + 1;
		offset_height[i] = offset;
		offset_skip[i] = random_index;
	}
	__syncthreads();

	for (int i = threadIdx.x; i < BLOCK_SIZE + 1024 - 100; i += BLOCK_SIZE) {
		random_index = i;
		for (int j = 0; j < 10; j++) {
			random_index += 6;
			random_index = offset_skip[random_index];
		}
		skip_index[i] = random_index;
	}
	__syncthreads();

	for (int i = threadIdx.x; i < BLOCK_SIZE + 1024 - 102; i += BLOCK_SIZE) {
		random_index = i + 2;
		int16_t terrain = next_int_unknown(random, &random_index, (FLOOR_LEVEL + 1) * 2);
		floor_terrain[i] = terrain;
		floor_skip[i] = random_index;
		if (terrain - 3 > FLOOR_LEVEL + CACTUS_HEIGHT + 1) {
			skip_first[i] = skip_index[random_index];
			skip_always[i] = skip_index[random_index];
		} else if (terrain - 3 > FLOOR_LEVEL + 1) {
			skip_first[i] = skip_index[random_index];
			skip_always[i] = 0;
		} else if (terrain + 3 <= FLOOR_LEVEL && terrain - 3 >= 0) {
			skip_first[i] = random_index + 60;
			skip_always[i] = random_index + 60;
		} else {
			skip_first[i] = 0;
			skip_always[i] = 0;
		}
	}
	__syncthreads();

	random_index = threadIdx.x;
	uint16_t best = 0;
	bool changed = false;
	int i = 0;
	for (; i < 10 && skip_first[random_index]; i++) {
		random_index = skip_first[random_index];
	}
	for (; i < 10; i++) {
		if (!changed && skip_first[random_index]) {
			random_index = skip_first[random_index];
			continue;
		}
		uint16_t bx = next(random, &random_index, 4) + 8;
		uint16_t bz = next(random, &random_index, 4) + 8;
		uint16_t initial = bx * 32 + bz;
		int16_t terrain;
		if (extract(heightmap, initial) == FLOOR_LEVEL) {
			if (skip_always[random_index - 2]) {
				random_index = skip_always[random_index - 2];
				continue;
			}
			terrain = floor_terrain[random_index - 2];
			random_index = floor_skip[random_index - 2];
		} else {
			terrain = next_int_unknown(random, &random_index, (extract(heightmap, initial) + 1) * 2);
			if (terrain + 3 <= FLOOR_LEVEL && terrain - 3 >= 0) {
				random_index += 60;
				continue;
			}
		}
		if (terrain - 3 > extract(heightmap, best) + 1) {
			random_index = skip_index[random_index];
			continue;
		}
		for (int j = 0; j < 10; j++) {
			int16_t bx = next(random, &random_index, 3) - next(random, &random_index, 3);
			int16_t by = next(random, &random_index, 2) - next(random, &random_index, 2);
			int16_t bz = next(random, &random_index, 3) - next(random, &random_index, 3);
			uint16_t xz = initial + bx * 32 + bz;
			int16_t y = (int16_t) terrain + by;
			if (y <= extract(heightmap, xz) && y >= 0) continue;
			uint8_t offset = offset_height[random_index];
			random_index = offset_skip[random_index];
			if (y != extract(heightmap, xz) + 1) continue;
			if (y == FLOOR_LEVEL + 1) {
				uint8_t mask = 0;
				if (bz != 0x00) mask |= extract(heightmap, xz - 1) - FLOOR_LEVEL;
				if (bz != 0x1F) mask |= extract(heightmap, xz + 1) - FLOOR_LEVEL;
				if (bx != 0x00) mask |= extract(heightmap, xz - 32) - FLOOR_LEVEL;
				if (bx != 0x1F) mask |= extract(heightmap, xz + 32) - FLOOR_LEVEL;
				if (mask) continue;
			}
			increase(heightmap, xz, offset);
			changed = true;
			if (extract(heightmap, xz) > extract(heightmap, best)) best = xz;
		}
	}
	if (extract(heightmap, best) - FLOOR_LEVEL >= CACTUS_HEIGHT) {
		out[atomicAdd((unsigned long long*) out_n, 1ULL)] = seed;
	}
}

unsigned long long block_add[BLOCK_SIZE + 1];
unsigned long long block_mul[BLOCK_SIZE + 1];
unsigned long long chunk_add[CHUNK_SIZE + 1];
unsigned long long chunk_mul[CHUNK_SIZE + 1];
unsigned long long offset = 0;
unsigned long long seed = 0;
unsigned long long total_seeds = 0;
time_t elapsed_chkpoint = 0;
std::mutex mutexcuda;
std::thread threads[1];

unsigned long long BEGIN;
unsigned long long BEGINOrig;
unsigned long long END;
int checkpoint_now;

struct checkpoint_vars {
unsigned long long offset;
time_t elapsed_chkpoint;
};
int32_t floor_level_host;
void run(int gpu_device)
{
	FILE* kaktseeds = fopen("kaktseeds.txt", "w+");
	unsigned long long *out;
	unsigned long long *out_n;
	hipSetDevice(gpu_device);
	hipMallocManaged(&out, GRID_SIZE * sizeof(*out));
	hipMallocManaged(&out_n, sizeof(*out_n));
	hipMemcpyToSymbol(HIP_SYMBOL(block_add_gpu), block_add, (BLOCK_SIZE + 1) * sizeof(*block_add));
	hipMemcpyToSymbol(HIP_SYMBOL(block_mul_gpu), block_mul, (BLOCK_SIZE + 1) * sizeof(*block_mul));
	hipMemcpyToSymbol(HIP_SYMBOL(chunk_add_gpu), chunk_add, (CHUNK_SIZE + 1) * sizeof(*chunk_add));
	hipMemcpyToSymbol(HIP_SYMBOL(chunk_mul_gpu), chunk_mul, (CHUNK_SIZE + 1) * sizeof(*chunk_mul));
	hipMemcpyToSymbol(HIP_SYMBOL(FLOOR_LEVEL), &floor_level_host, sizeof(int32_t));
	while (true) {
		*out_n = 0;
		{
			if (offset >= END) break;
			unsigned long long seed_gpu = (seed * RNG_MUL + RNG_ADD) & RNG_MASK;
			crack<<<CHUNK_SIZE, BLOCK_SIZE>>>(seed_gpu, out, out_n);
			offset += GRID_SIZE;
			seed = (seed * chunk_mul[CHUNK_SIZE] + chunk_add[CHUNK_SIZE]) & RNG_MASK;
		}
		hipDeviceSynchronize();
		{
			total_seeds += *out_n;
			for (unsigned long long i = 0; i < *out_n; i++){
				fprintf(kaktseeds,"s: %llu,\n", out[i]);
			}
			fflush(kaktseeds);
		}
	}
	fclose(kaktseeds);
	hipFree(out_n);
	hipFree(out);
}

int main(int argc, char *argv[])
{
	#ifdef BOINC
	BOINC_OPTIONS options;

	boinc_options_defaults(options);
	options.normal_thread_priority = true;
	boinc_init_options(&options);
	#endif
	
	block_add[0] = 0;
	block_mul[0] = 1;
	for (unsigned long long i = 0; i < BLOCK_SIZE; i++) {
		block_add[i + 1] = (block_add[i] * RNG_MUL + RNG_ADD) & RNG_MASK;
		block_mul[i + 1] = (block_mul[i] * RNG_MUL) & RNG_MASK;
	}

	chunk_add[0] = 0;
	chunk_mul[0] = 1;
	for (unsigned long long i = 0; i < CHUNK_SIZE; i++) {
		chunk_add[i + 1] = (chunk_add[i] * block_mul[BLOCK_SIZE] + block_add[BLOCK_SIZE]) & RNG_MASK;
		chunk_mul[i + 1] = (chunk_mul[i] * block_mul[BLOCK_SIZE]) & RNG_MASK;
	}
	
	int gpu_device = 0;
	for (int i = 1; i < argc; i += 2) {
		const char *param = argv[i];
		if (strcmp(param, "-d") == 0 || strcmp(param, "--device") == 0) {
			gpu_device = atoi(argv[i + 1]);
		} else if (strcmp(param, "-s") == 0 || strcmp(param, "--start") == 0) {
			sscanf(argv[i + 1], "%llu", &BEGIN);
		} else if (strcmp(param, "-e") == 0 || strcmp(param, "--end") == 0) {
			sscanf(argv[i + 1], "%llu", &END);
		} else if (strcmp(param, "-h") == 0 || strcmp(param, "--height") == 0){
			sscanf(argv[i + 1], "%llu", &floor_level_host);
		} else {
			fprintf(stderr,"Unknown parameter: %s\n", param);
		}
	}
	BEGINOrig = BEGIN;

	FILE *checkpoint_data = boinc_fopen("kaktpoint.txt", "rb");

	if (!checkpoint_data) {
		fprintf(stderr,"No checkpoint to load\n");
	} else {
		#ifdef BOINC
		boinc_begin_critical_section();
		#endif 

		struct checkpoint_vars data_store;
		fread(&data_store, sizeof(data_store), 1, checkpoint_data);

		BEGIN = data_store.offset;
		elapsed_chkpoint = data_store.elapsed_chkpoint;

		fprintf(stderr,"Checkpoint loaded, task time %d s, seed pos: %llu\n", elapsed_chkpoint, BEGIN);
		fclose(checkpoint_data);
		
		#ifdef BOINC
		boinc_end_critical_section();
		#endif
	}

	for (; offset + GRID_SIZE <= BEGIN; offset += GRID_SIZE)
		seed = (seed * chunk_mul[CHUNK_SIZE] + chunk_add[CHUNK_SIZE]) & RNG_MASK;
	for (; offset + 1 <= BEGIN; offset += 1)
		seed = (seed * RNG_MUL + RNG_ADD) & RNG_MASK;

	#ifdef BOINC
	APP_INIT_DATA aid;
	boinc_get_init_data(aid);
	
	if (aid.gpu_device_num >= 0) {
		gpu_device = aid.gpu_device_num;
		fprintf(stderr,"boinc gpu %i gpuindex: %i \n", aid.gpu_device_num, gpu_device);
		} else {
		fprintf(stderr,"stndalone gpuindex %i \n", gpu_device);
	}
	#endif
	threads[0] = std::thread(run, gpu_device);

	checkpoint_now = 0;
	time_t start_time = time(NULL);
	while (offset < END) {
		using namespace std::chrono_literals;
		std::this_thread::sleep_for(1s);
		time_t elapsed = time(NULL) - start_time;
		unsigned long long count = offset - BEGIN;
		double frac = (double) count / (double) (END - BEGIN);
		
		#ifdef BOINC
		boinc_fraction_done(frac);
		#endif
		
		checkpoint_now++;

		if (checkpoint_now >= 30 || boinc_time_to_checkpoint() ){  // 30 for 30 secs before checkpoint
		
		#ifdef BOINC
		boinc_begin_critical_section(); // Boinc should not interrupt this
		#endif
		
		// Checkpointing section below
			boinc_delete_file("kaktpoint.txt"); // Don't touch, same func as normal fdel
			FILE *checkpoint_data = boinc_fopen("kaktpoint.txt", "wb");

			struct checkpoint_vars data_store;
			data_store.offset = offset;
			data_store.elapsed_chkpoint = elapsed_chkpoint + elapsed;

			fwrite(&data_store, sizeof(data_store), 1, checkpoint_data);

			fclose(checkpoint_data);
			checkpoint_now=0;

		#ifdef BOINC
		boinc_end_critical_section();
		boinc_checkpoint_completed(); // Checkpointing completed
		#endif
		}
	}
	
	#ifdef BOINC
	boinc_begin_critical_section();
	#endif

	for (std::thread& thread : threads)
		thread.join();

	time_t elapsed = time(NULL) - start_time;
	unsigned long long count = offset - BEGIN;
	double done = (double) count / 1000000.0;
	double speed = done / (double) elapsed;

	fprintf(stderr, "\nSpeed: %.2lfm/s\n", speed );
        fprintf(stderr, "Done\n");
	fprintf(stderr, "Processed: %llu seeds in %.2lfs seconds\n", END - BEGINOrig, (double) elapsed_chkpoint + (double) elapsed );

	#ifdef BOINC
	boinc_end_critical_section();
	#endif

	boinc_finish(0);
}